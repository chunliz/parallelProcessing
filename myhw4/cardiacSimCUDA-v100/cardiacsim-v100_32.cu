/* 
 * Solves the Panfilov model using an explicit numerical scheme.
 * Based on code orginally provided by Xing Cai, Simula Research Laboratory 
 * and reimplementation by Scott B. Baden, UCSD
 * 
 * Modified and  restructured by Didem Unat, Koc University
 *
 */
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"

#define BLOCK_WIDTH 32

using namespace std;


// Utilities
// 

// Timer
// Make successive calls and take a difference to get the elapsed time.
static const double kMicro = 1.0e-6;
double getTime()
{
    struct timeval TV;
    struct timezone TZ;

    const int RC = gettimeofday(&TV, &TZ);
    if(RC == -1) {
            cerr << "ERROR: Bad call to gettimeofday" << endl;
            return(-1);
    }

    return( ((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec) );

}  // end getTime()

/*
// Allocate a 2D array
double **alloc2D(int m,int n){
   double **E;
   int nx=n, ny=m;
   E = (double**)malloc(sizeof(double*)*ny + sizeof(double)*nx*ny);
   assert(E);
   int j;
   for(j=0;j<ny;j++) 
     E[j] = (double*)(E+ny) + j*nx;
   return(E);
}
*/
    
// Reports statistics about the computation
// These values should not vary (except to within roundoff)
// when we use different numbers of  processes to solve the problem
 double stats(double *E, int m, int n, double *_mx){
     double mx = -1;
     double l2norm = 0;
     int i, j;
     for (j=1; j<=m; j++)
       for (i=1; i<=n; i++) {
       l2norm += E[j*(n+2)+i]*E[j*(n+2)+i];
       if (E[j*(n+2)+i] > mx)
           mx = E[j*(n+2)+i];
      }
     *_mx = mx;
     l2norm /= (double) ((m)*(n));
     l2norm = sqrt(l2norm);
     return l2norm;
 }

// External functions
extern "C" {
    void splot(double *E, double T, int niter, int m, int n);
}
void cmdLine(int argc, char *argv[], float& T, int& n, int& px, int& py, int& plot_freq, int& no_comm, int&num_threads);

__global__ void boundaryKernal(double *E_prev, int n, int m){
  
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if ((col == 0) && (row < m))
        E_prev[(row+1)*(n+2)] = E_prev[(row+1)*(n+2)+2];
    if ((col == (n-1)) && (row < m))
        E_prev[(row+1)*(n+2)+n+1] = E_prev[(row+1)*(n+2)+n-1];
    if ((row == 0) && (col < n))
        E_prev[col+1] = E_prev[col+1+2*(n+2)];
    if ((row == (m-1)) && (col < n))
        E_prev[(m+1)*(n+2)+col+1] = E_prev[(m-1)*(n+2)+col+1];
    __syncthreads();
}


__global__ void PDEKernal(double *E, double *E_prev, int n, int m, double alpha){

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if((row < m) && (col < n)){
    
        E[(row+1)*(n+2)+col+1] = E_prev[(row+1)*(n+2)+col+1]+alpha*(E_prev[(row+1)*(n+2)+col+2]+E_prev[(row+1)*(n+2)+col]-4*E_prev[(row+1)*(n+2)+col+1]+E_prev[(row+2)*(n+2)+col+1]+E_prev[row*(n+2)+col+1]);
    }
    __syncthreads();

}

__global__ void ODEKernal1(double *E, double *R, int n, int m, double kk, double dt, double a){

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if((row < m) && (col < n)){
    
        E[(row+1)*(n+2)+col+1] = E[(row+1)*(n+2)+col+1] -dt*(kk* E[(row+1)*(n+2)+col+1]*(E[(row+1)*(n+2)+col+1] - a)*(E[(row+1)*(n+2)+col+1]-1)+ E[(row+1)*(n+2)+col+1] *R[(row+1)*(n+2)+col+1]);
    }
    __syncthreads();
}

__global__ void ODEKernal2(double *E, double *R, int n, int m, double kk, double dt, double b,  double epsilon, double M1, double M2){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if((row < m) && (col < n)){
        R[(row+1)*(n+2)+col+1] = R[(row+1)*(n+2)+col+1] + dt*(epsilon+M1* R[(row+1)*(n+2)+col+1]/( E[(row+1)*(n+2)+col+1]+M2))*(-R[(row+1)*(n+2)+col+1]-kk* E[(row+1)*(n+2)+col+1]*(E[(row+1)*(n+2)+col+1]-b-1));
        
    }
    __syncthreads();
}

__global__ void swapKernal(double *E, double *E_prev, int n, int m){

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    double tmp = 0.0;
    if ((row < m) && (col < n)){
        tmp = E[(row+1)*(n+2)+col+1];
        E[(row+1)*(n+2)+col+1] = E_prev[(row+1)*(n+2)+col+1];
        E_prev[(row+1)*(n+2)+col+1] = tmp;
    }
    __syncthreads();
    
}


/*
void simulate (double* E,  double* E_prev,double* R,
	       const double alpha, const int n, const int m, const double kk,
	       const double dt, const double a, const double epsilon,
	       const double M1,const double  M2, const double b)
{
  int i, j; 
    /* 
     * Copy data from boundary of the computational box 
     * to the padding region, set up for differencing
     * on the boundary of the computational box
     * Using mirror boundaries
     */
/*

    for (j=1; j<=m; j++) 
      E_prev[j*(m+2)] = E_prev[j*(m+2)+2];
    for (j=1; j<=m; j++) 
      E_prev[j*(m+2)+n+1] = E_prev[j*(m+2)+n-1];
    
    for (i=1; i<=n; i++) 
      E_prev[i] = E_prev[2*(m+2)+i];
    for (i=1; i<=n; i++) 
      E_prev[(m+1)*(m+2)+i] = E_prev[(m-1)*(m+2)+i];

    
    // Solve for the excitation, the PDE
    for (j=1; j<=m; j++){
      for (i=1; i<=n; i++) {
    E[j*(m+2)+i] = E_prev[j*(m+2)+i]+alpha*(E_prev[j*(m+2)+i+1]+E_prev[j*(m+2)+i-1]-4*E_prev[j*(m+2)+i]+E_prev[(j+1)*(m+2)+i]+E_prev[(j-1)*(m+2)+i]);
      }
    }
    
    /* 
     * Solve the ODE, advancing excitation and recovery to the
     *     next timtestep
     */

/*
    for (j=1; j<=m; j++){
      for (i=1; i<=n; i++)
    E[j*(m+2)+i] = E[j*(m+2)+i] -dt*(kk* E[j*(m+2)+i]*(E[j*(m+2)+i] - a)*(E[j*(m+2)+i]-1)+ E[j*(m+2)+i] *R[j*(m+2)+i]);
    }
    
    for (j=1; j<=m; j++){
      for (i=1; i<=n; i++)
    R[j*(m+2)+i] = R[j*(m+2)+i] + dt*(epsilon+M1* R[j*(m+2)+i]/( E[j*(m+2)+i]+M2))*(-R[j*(m+2)+i]-kk* E[j*(m+2)+i]*(E[j*(m+2)+i]-b-1));
    }
    
}

*/

// Main program
int main (int argc, char** argv)
{
  /*
   *  Solution arrays
   *   E is the "Excitation" variable, a voltage
   *   R is the "Recovery" variable
   *   E_prev is the Excitation variable for the previous timestep,
   *      and is used in time integration
   */
  double *E, *R, *E_prev;
  
  // Various constants - these definitions shouldn't change
  const double a=0.1, b=0.1, kk=8.0, M1= 0.07, M2=0.3, epsilon=0.01, d=5e-5;
  
  float T=1000.0;
  int m=200,n=200;
  int plot_freq = 0;
  int px = 1, py = 1;
  int no_comm = 0;
  int num_threads=1; 

  cmdLine( argc, argv, T, n,px, py, plot_freq, no_comm, num_threads);
  m = n;  
  int size=(m+2)*(n+2);

  // Allocate contiguous memory for solution arrays
  // The computational box is defined on [1:m+1,1:n+1]
  // We pad the arrays in order to facilitate differencing on the 
  // boundaries of the computation box

  E=(double *)malloc(size*sizeof(double));
  E_prev=(double *)malloc(size*sizeof(double));
  R=(double *)malloc(size*sizeof(double));
  
  int i,j;
  // Initialization
  for (j=1; j<=m; j++)
    for (i=1; i<=n; i++)
      E_prev[j*(n+2)+i] = R[j*(n+2)+i] = 0;
  
  for (j=1; j<=m; j++)
    for (i=n/2+1; i<=n; i++)
      E_prev[j*(n+2)+i] = 1.0;
  
  for (j=m/2+1; j<=m; j++)
    for (i=1; i<=n; i++)
      R[j*(n+2)+i] = 1.0;
  
  double dx = 1.0/n;

  // For time integration, these values shouldn't change 
  double rp= kk*(b+1)*(b+1)/4;
  double dte=(dx*dx)/(d*4+((dx*dx))*(rp+kk));
  double dtr=1/(epsilon+((M1/M2)*rp));
  double dt = (dte<dtr) ? 0.95*dte : 0.95*dtr;
  double alpha = d*dt/(dx*dx);

  cout << "Grid Size       : " << n << endl; 
  cout << "Duration of Sim : " << T << endl; 
  cout << "Time step dt    : " << dt << endl; 
  cout << "Process geometry: " << px << " x " << py << endl;
  if (no_comm)
    cout << "Communication   : DISABLED" << endl;
  
  cout << endl;
  
  // Start the timer
  double t0 = getTime();
  
 
  // Simulated time is different from the integer timestep number
  // Simulated time
  double t = 0.0;
  // Integer timestep number
  int niter=0;

  double *d_E, *d_E_prev, *d_R;

  hipMalloc((void **) &d_E, size*sizeof(double));
  hipMemcpy(d_E, E, size*sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void **) &d_E_prev, size*sizeof(double));
  hipMemcpy(d_E_prev, E_prev, size*sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void **) &d_R, size*sizeof(double));
  hipMemcpy(d_R, R, size*sizeof(double), hipMemcpyHostToDevice);

  int tnx=n/BLOCK_WIDTH;
  int tny=m/BLOCK_WIDTH;

  if(n%BLOCK_WIDTH) tnx++;
  if(m%BLOCK_WIDTH) tny++;

  dim3 dimGrid(tnx,tny,1);
  dim3 dimBlock(BLOCK_WIDTH,BLOCK_WIDTH,1);

  double t1 = getTime();
  
  while (t<T) {
    
    t += dt;
    niter++;

    boundaryKernal<<<dimGrid,dimBlock>>>(d_E_prev, n, m);
    hipDeviceSynchronize();

    PDEKernal<<<dimGrid,dimBlock>>>(d_E, d_E_prev, n, m, alpha);
    hipDeviceSynchronize();

    ODEKernal1<<<dimGrid,dimBlock>>>(d_E, d_R, n, m, kk, dt, a);
    hipDeviceSynchronize();

    ODEKernal2<<<dimGrid,dimBlock>>>(d_E, d_R, n, m, kk, dt, b, epsilon, M1, M2);
    hipDeviceSynchronize();
 
    //simulate(E, E_prev, R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);

    swapKernal<<<dimGrid,dimBlock>>>(d_E, d_E_prev, n, m);
    hipDeviceSynchronize();
    
    //swap current E with previous E
    //double *tmp = E; E = E_prev; E_prev = tmp;
    
    if (plot_freq){

      int k = (int)(t/plot_freq);
      if ((t - k * plot_freq) < dt){
        hipMemcpy(E, d_E, size*sizeof(double), hipMemcpyDeviceToHost);
	splot(E,t,niter,m+2,n+2);
      }
    }
  }//end of while loop

  double time_elapsed1 = getTime() - t1;

  hipMemcpy(E, d_E, size*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(E_prev, d_E_prev, size*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(R, d_R, size*sizeof(double), hipMemcpyDeviceToHost);

  hipFree(d_E);
  hipFree(d_E_prev);
  hipFree(d_R);

  double time_elapsed = getTime() - t0;

  double Gflops = (double)(niter * (1E-9 * n * n ) * 28.0) / time_elapsed ;
  double BW = (double)(niter * 1E-9 * (n * n * sizeof(double) * 4.0  ))/time_elapsed;
  cout << "BLOCK SIZE                  : " << BLOCK_WIDTH << endl;
  cout << "Number of Iterations        : " << niter << endl;
  cout << "Elapsed Time (sec)          : " << time_elapsed << endl;
  cout << "Time (no data transfer)     : " << time_elapsed1 <<endl;
  cout << "Sustained Gflops Rate       : " << Gflops << endl; 
  cout << "Sustained Bandwidth (GB/sec): " << BW << endl << endl;
  cout << "************************************************************" <<endl;
  cout << "************************************************************" <<endl;
  cout << "************************************************************" <<endl;

  //for(i=1;i<=m;i++) printf("E[%d,%d]:%f\n",i,i,E_prev[i*(m+2)+i]);

  double mx;
  double l2norm = stats(E_prev,m,n,&mx);
  cout << "Max: " << mx <<  " L2norm: "<< l2norm << endl;

  if (plot_freq){
    cout << "\n\nEnter any input to close the program and the plot..." << endl;
    getchar();
  }
  
  free (E);
  free (E_prev);
  free (R);
  
  return 0;
}
